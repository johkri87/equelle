#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include <opm/common/ErrorMacros.hpp>

#include <thrust/device_vector.h>
#include <thrust/detail/raw_pointer_cast.h>

#include "DeviceGrid.hpp"
#include "wrapDeviceGrid.hpp"
#include "CollOfScalar.hpp"
#include "CollOfIndices.hpp"
#include "equelleTypedefs.hpp"
#include "device_functions.cuh"

using namespace equelleCUDA;

// --------------------------------------------
//                     EXTEND
// --------------------------------------------

CollOfScalar wrapDeviceGrid::extendToFull( const CollOfScalar& in_data,
					   const thrust::device_vector<int>& from_set,
					   const int full_size) {
    // setup how many threads/blocks we need:
    kernelSetup s(full_size);
    
    // create a vector of size number_of_faces_:
    CudaArray val(full_size);
    const int* from_ptr = thrust::raw_pointer_cast( &from_set[0]);

    // Extend values
    wrapDeviceGrid::extendToFullKernel_step1<<<s.grid, s.block>>>( val.data(), full_size );
    wrapDeviceGrid::extendToFullKernel_step2<<<s.grid, s.block>>>( val.data(), from_ptr,
                                                                   from_set.size(), in_data.data());
    hipDeviceSynchronize();
    if (in_data.useAutoDiff() ) {
       CudaMatrix inMat(std::move(in_data.derivative()));
       CudaMatrix der(full_size, inMat.cols(), inMat.nnz());
       kernelSetup sNnz(inMat.nnz());
       wrapDeviceGrid::extendToFullKernel_step1<<<sNnz.grid, sNnz.block>>>( der.csrVal(), inMat.nnz() );
       hipDeviceSynchronize();

       kernelSetup sRptr(from_set.size());
       wrapDeviceGrid::extendToFullKernel_step1<<<s.grid, s.block>>>( der.csrRowPtr(), full_size+1 );
       hipDeviceSynchronize();
       wrapDeviceGrid::extendToFullKernel_step2<<<sRptr.grid, sRptr.block>>>( der.csrRowPtr(), from_ptr,
                                                                   from_set.size(), inMat.csrRowPtr());
       hipDeviceSynchronize();
        wrapDeviceGrid::extendToFullKernel_buildRowPtr<<<sRptr.grid, sRptr.block>>>( der.csrRowPtr(), from_ptr,
                                                                   from_set.size(), inMat.csrRowPtr(), full_size);
       hipError_t error = hipMemcpy(der.csrColInd(),inMat.csrColInd(), inMat.nnz()*sizeof(int),hipMemcpyDeviceToDevice);
       if (error != hipSuccess){
            std::cout << "Memcpy csrColInd failed in wrapDevideGrid::extendToFull." << std::endl;
       }
       error = hipMemcpy(der.csrVal(),inMat.csrVal(), inMat.nnz()*sizeof(double),hipMemcpyDeviceToDevice);
       if (error != hipSuccess){
            std::cout << "Memcpy csrVal() failed in wrapDevideGrid::extendToFull." << std::endl;
       }
       hipDeviceSynchronize();

       // Old code
       // CudaMatrix extendMatrix = CudaMatrix(from_set, full_size).transpose();
       // return CollOfScalar(val, extendMatrix * in_data.derivative());
       return CollOfScalar(val, der);
    }
    return CollOfScalar(val);
}






CollOfScalar wrapDeviceGrid::extendToSubset( const CollOfScalar& inData,
					     const thrust::device_vector<int>& from_set,
					     const thrust::device_vector<int>& to_set,
					     const int full_size) {
    CollOfScalar temp_full = extendToFull( inData, from_set, full_size);
    return onFromFull(temp_full, to_set);

}


// Set all values to 0
__global__ void wrapDeviceGrid::extendToFullKernel_step1( double* outData,
							  const int out_size)
{
    const int outIndex = myID();
    if ( outIndex < out_size ) {
	   outData[outIndex] = 0.0;
    }
}

// Set all values to 0
__global__ void wrapDeviceGrid::extendToFullKernel_step1( int* outData,
                              const int out_size)
{
    const int outIndex = myID();
    if ( outIndex < out_size ) {
       outData[outIndex] = 0;
    }
}


// Place old values in the new domain
__global__ void wrapDeviceGrid::extendToFullKernel_step2( double* outData,
							  const int* from_set,
							  const int from_size,
							  const double* inData)
{
    //
    //      This kernel is sensitive to a race condition!
    //      Each thread with outIndex < from_size performs 2 write operations,
    //      but not to the same memory.
    //      Hence, the we can have a kernel with 
    //          outIndex = 3;
    //	  outData[3] = 0;
    //	  from_set[3] = 1000;
    //	  outData[1000] = 3.14;
    //     And then another block starting a bit later with
    //         outIndex = 1000;
    //	 outData[1000] = 0; // overwriting outIndex(3)'s correct value
    //
    //	 Only way to sync between blocks is to call seperate kernels!
    //

    const int outIndex = myID();
    if ( outIndex < from_size ) {
	outData[from_set[outIndex]] = inData[outIndex];
    }
}

// Place old values in the new domain
__global__ void wrapDeviceGrid::extendToFullKernel_step2( int* outData,
                              const int* from_set,
                              const int from_size,
                              const int* inData)
{

    const int outIndex = myID();
    if ( outIndex < from_size ) {
        outData[from_set[outIndex]+1] = inData[outIndex+1];
    }
}

// Build the csr pointer by filling in values for the empty areas in the new extended domain.
__global__ void wrapDeviceGrid::extendToFullKernel_buildRowPtr( int* outData,
                              const int* from_set,
                              const int from_size,
                              const int* inData,
                              const int full_size)
{
    const int outIndex = myID();
    if ( outIndex < from_size-1 ) {
        int diff = from_set[outIndex+1] - from_set[outIndex];
        for( int i = 1; i < diff+1; i++ ){
            outData[from_set[outIndex]+i] = inData[outIndex+1];
        }
        if ( outIndex == from_size-2 ){
            int lastDiff = (full_size-from_set[from_size-1]);
            for( int i = 1; i <= lastDiff; i++ ){
                outData[from_set[from_size-1]+i] = inData[from_size];
            }
        }
    }
}


// ------------------------------------------------
//                ON for CollOfScalar
// ------------------------------------------------

CollOfScalar wrapDeviceGrid::onFromFull( const CollOfScalar& inData,
					 const thrust::device_vector<int>& to_set ) {

    // inData is a full set, so position is its index
    // to_set is indices which we get the input from.
    // out will be same size as to_set.

    // setup how many threads/blocks we need:
    kernelSetup s(to_set.size());

    // Create the output vector:
    CudaArray val(to_set.size());
    const int* to_set_ptr = thrust::raw_pointer_cast( &to_set[0] );
    wrapDeviceGrid::onFromFullKernel<<<s.grid, s.block>>>(val.data(),
							  to_set_ptr,
							  to_set.size(),
							  inData.data());
    if ( inData.useAutoDiff() ) {
	CudaMatrix onMatrix(to_set, inData.size());
	return CollOfScalar(val, onMatrix * inData.derivative());
    }
    else { // no AutoDiff
	return CollOfScalar(val);
    }    

    // Use the matrix and find the result from Matrix-vector multiplication
    //CudaMatrix onMatrix(to_set, inData.size());
    //return onMatrix * inData;
}

CollOfScalar wrapDeviceGrid::onFromSubset( const CollOfScalar& inData,
					   const thrust::device_vector<int>& from_set,
					   const thrust::device_vector<int>& to_set,
					   const int full_size) {
    
    CollOfScalar temp_full = extendToFull(inData, from_set, full_size);
    return onFromFull(temp_full, to_set);
}



__global__ void wrapDeviceGrid::onFromFullKernel( double* outData,
						  const int* to_set,
						  const int to_size,
						  const double* inData)
{
    const int toIndex = myID();
    if ( toIndex < to_size ) {
	outData[toIndex] = inData[to_set[toIndex]];
    }
}
						  

// -----------------------------------------------
//              ON for CollOfIndices
// -----------------------------------------------

thrust::device_vector<int> wrapDeviceGrid::onFromFullIndices( const thrust::device_vector<int>& inData,
							      const thrust::device_vector<int>& to_set ) {

    // inData is a full set, so position is its index
    // to_set is indices which we get the input from.
    // out will be same size as to_set.

    // setup how many threads/blocks we need:
    kernelSetup s(to_set.size());

    // Create the output vector:
    thrust::device_vector<int> out(to_set.size());
    const int* to_set_ptr = thrust::raw_pointer_cast( &to_set[0] );
    const int* inData_ptr = thrust::raw_pointer_cast( &inData[0] );
    int* out_ptr = thrust::raw_pointer_cast( &out[0] );
    wrapDeviceGrid::onFromFullKernelIndices<<<s.grid, s.block>>>(out_ptr,
								 to_set_ptr,
								 to_set.size(),
								 inData_ptr);
    return out;
}



thrust::device_vector<int> wrapDeviceGrid::onFromSubsetIndices( const thrust::device_vector<int>& inData,
								const thrust::device_vector<int>& from_set,
								const thrust::device_vector<int>& to_set,
								const int full_size) {
    
    thrust::device_vector<int> temp_full = extendToFullIndices(inData, from_set, full_size);
    return onFromFullIndices(temp_full, to_set);
}




__global__ void wrapDeviceGrid::onFromFullKernelIndices( int* outData,
							 const int* to_set,
							 const int to_size,
							 const int* inData)
{
    const int toIndex = myID();
    if ( toIndex < to_size ) {
	outData[toIndex] = inData[to_set[toIndex]];
    }
}


thrust::device_vector<int> wrapDeviceGrid::extendToFullIndices( const thrust::device_vector<int>& in_data,
								const thrust::device_vector<int>& from_set,
								const int full_size) {
    // setup how many threads/blocks we need:
    kernelSetup s(full_size);

    // create a vector of size number_of_faces_:
    thrust::device_vector<int> out(full_size);
    int* out_ptr = thrust::raw_pointer_cast( &out[0] );
    const int* in_data_ptr = thrust::raw_pointer_cast( &in_data[0] );
    const int* from_ptr = thrust::raw_pointer_cast( &from_set[0]);
    wrapDeviceGrid::extendToFullKernelIndices_step1<<<s.grid, s.block>>>( out_ptr,
									  full_size);
    wrapDeviceGrid::extendToFullKernelIndices_step2<<<s.grid, s.block>>>( out_ptr,
									  from_ptr,
									  from_set.size(),
									  in_data_ptr);
    
      
    return out;
}



// EXTEND TO FULL FOR INDICES DONE IN 2 STEPS

__global__ void wrapDeviceGrid::extendToFullKernelIndices_step1( int* outData,
								 const int full_size)
{
    const int outIndex = myID();
    if ( outIndex < full_size) {
	outData[outIndex] = 0;
    }
}

__global__ void wrapDeviceGrid::extendToFullKernelIndices_step2( int* outData,
								 const int* from_set,
								 const int from_size,
								 const int* inData)
{
    const int outIndex = myID();
    if ( outIndex < from_size ) {
	outData[from_set[outIndex]] = inData[outIndex];
    }
}

