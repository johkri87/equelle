#include "hip/hip_runtime.h"
#include "common_kernels.cuh"
#include "device_functions.cuh"

using namespace equelleCUDA;

__global__ void EquelleKernels::multiplication_kernel(double* out, const double* rhs, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = out[index] * rhs[index];
    }
}

__global__ void EquelleKernels::multiplication_kernel(double* out, const double rhs, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = out[index] * rhs;
    }
}

__global__ void EquelleKernels::addition_kernel(double* out, const double* rhs, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = out[index] + rhs[index];
    }
}

__global__ void EquelleKernels::division_kernel(double* out, const double* rhs, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = out[index] / rhs[index];
    }
}

__global__ void EquelleKernels::division_kernel(const double lhs, double* out, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = lhs / out[index];
    }
}

__global__ void EquelleKernels::division_kernel(double* out, const double rhs, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = out[index] / rhs;
    }
}

__global__ void EquelleKernels::negate_kernel(double* out, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = -out[index];
    }
}

__global__ void EquelleKernels::square_kernel(double* out, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = out[index] * out[index];
    }
}

__global__ void EquelleKernels::sqrt_kernel(double* out, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = __dsqrt_rn(out[index]);
    }
}

__global__ void EquelleKernels::reciprocal_kernel(double* out, const int size) {
    const int index = myID();
    if ( index < size ) {
        out[index] = __drcp_rn(out[index]);
    }
}

__global__ void EquelleKernels::abs_kernel(double* out, const int size) {
    const int i = myID();
    if ( i < size ) {
        out[i] = fabs(out[i]);
    }
}