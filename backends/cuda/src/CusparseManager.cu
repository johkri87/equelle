#include "CusparseManager.hpp"
#include <time.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <hipblas.h>

using namespace equelleCUDA;

CusparseManager::CusparseManager()
    : buffer_(NULL),
      currentBufferSize_(0)
{
    std::cout << "CusparseManager constructed." << std::endl;
    // Set up cuSPARSE
    hipsparseCreate(&cusparseHandle_);
    hipsparseSetPointerMode(cusparseHandle_, HIPSPARSE_POINTER_MODE_HOST);
    hipsparseCreateCsrgemm2Info(&gemm2Info_);
}

CusparseManager::~CusparseManager()
{
    std::cout << "CusparseManager destroyed." << std::endl;

    if (buffer_) {
        hipFree(buffer_);
    }
    hipsparseDestroy(cusparseHandle_);
    hipsparseDestroyCsrgemm2Info(gemm2Info_);
}

/// Using the Meyers singleton pattern.
CusparseManager& CusparseManager::instance()
{
    static CusparseManager s;
    return s;
}


CudaMatrix CusparseManager::matrixMultiply(const CudaMatrix& lhs, const CudaMatrix& rhs)
{
    return instance().gemm(lhs, rhs);
}

// gemm2 is slower then gemm for simple Matrix-Matrix multiplication.
// However, we keep this for testing and profiling purposes.
CudaMatrix CusparseManager::matrixMultiply2(const CudaMatrix& A, const CudaMatrix& B)
{
    double alpha = 1.0;
    return instance().gemm2(A, B, CudaMatrix(), &alpha, NULL);
}

CudaMatrix CusparseManager::matrixAddition(const CudaMatrix& lhs, const CudaMatrix& rhs)
{
    double alpha = 1.0;
    double beta = 1.0;
    return instance().geam(lhs, rhs, &alpha, &beta);
}

CudaMatrix CusparseManager::matrixSubtraction(const CudaMatrix& lhs, const CudaMatrix& rhs)
{
    double alpha = 1.0;
    double beta = -1.0;
    return instance().geam(lhs, rhs, &alpha, &beta);
}

// gemm2, as opposed to gemm, does not call hipFree implicitly.
CudaMatrix CusparseManager::gemm2(const CudaMatrix& A, const CudaMatrix& B, const CudaMatrix& C, const double* alpha, const double* beta)
{
    CudaMatrix out;
    int innerSize = out.confirmMultSize(A, B);

    // Allocate buffer
    size_t newBufferSize;
    out.sparseStatus_ = hipsparseDcsrgemm2_bufferSizeExt(cusparseHandle_, out.rows_, out.cols_, innerSize, alpha,
                                     A.description_, A.nnz_, A.csrRowPtr_, A.csrColInd_,
                                     B.description_, B.nnz_, B.csrRowPtr_, B.csrColInd_,
                                     beta,
                                     C.description_, C.nnz_, C.csrRowPtr_, C.csrColInd_,
                                     gemm2Info_, &newBufferSize);
    out.checkError_("hipsparseDcsrgemm2_bufferSizeExt() in CusparseManager::gemm2()");
    if (newBufferSize > currentBufferSize_) {
        if (buffer_ != NULL) {
            out.cudaStatus_ = hipFree(buffer_);
            out.checkError_("hipsparseDcsrgemm2() in CusparseManager::gemm2()");
        }
        out.cudaStatus_ = hipMalloc(&buffer_, newBufferSize);
        out.checkError_("hipMalloc(&buffer_, newBufferSize) in CusparseManager::gemm2()");
        currentBufferSize_ = newBufferSize;
    }

    // Allocate row pointer
    out.cudaStatus_ = hipMalloc((void**)&out.csrRowPtr_, sizeof(int)*(out.rows_+1));
    out.checkError_("hipMalloc((void**)&out.csrRowPtr_, sizeof(int)*(out.rows_+1)) in CusparseManager::gemm2()");

    // Compute NNZ
    int* nnzTotalDevHostPtr = &out.nnz_;
    out.sparseStatus_ = hipsparseXcsrgemm2Nnz(cusparseHandle_,
                         out.rows_, out.cols_, innerSize,
                         A.description_, A.nnz_, A.csrRowPtr_, A.csrColInd_,
                         B.description_, B.nnz_, B.csrRowPtr_, B.csrColInd_,
                         C.description_, C.nnz_, C.csrRowPtr_, C.csrColInd_,
                         out.description_, out.csrRowPtr_,
                         nnzTotalDevHostPtr, gemm2Info_, buffer_);
    out.checkError_("hipsparseXcsrgemm2Nnz() in CusparseManager::gemm2()");
    if (NULL != nnzTotalDevHostPtr) {
        out.nnz_ = *nnzTotalDevHostPtr;
    } else
    {
        int baseC;
        out.cudaStatus_ = hipMemcpy(&out.nnz_, out.csrRowPtr_+out.rows_, sizeof(int), hipMemcpyDeviceToHost);
        out.checkError_("hipMemcpy(&out.nnz_, out.csrRowPtr_+out.rows_, sizeof(int), hipMemcpyDeviceToHost) in CusparseManager::gemm2()");
        out.cudaStatus_ = hipMemcpy(&baseC, out.csrRowPtr_, sizeof(int), hipMemcpyDeviceToHost);
        out.checkError_("hipMemcpy(&baseC, out.csrRowPtr_, sizeof(int), hipMemcpyDeviceToHost) in CusparseManager::gemm2()");
        out.nnz_ -= baseC;
    }

    // Allocate memory for output matrix
    out.cudaStatus_ = hipMalloc((void**)&out.csrColInd_, sizeof(int)*out.nnz_);
    out.checkError_("hipMalloc((void**)&out.csrColInd_, sizeof(int)*out.nnz_) in CusparseManager::gemm2()");
    out.cudaStatus_ = hipMalloc((void**)&out.csrVal_, sizeof(double)*out.nnz_);
    out.checkError_("hipMalloc((void**)&out.csrVal_, sizeof(double)*out.nnz_) in CusparseManager::gemm2()");
    
    // Perform the gemm2 operation
    // D = alpha ∗ A ∗ B + beta ∗ C
    out.sparseStatus_ = hipsparseDcsrgemm2(cusparseHandle_, out.rows_, out.cols_, innerSize, alpha, 
                      A.description_, A.nnz_, A.csrVal_, A.csrRowPtr_, A.csrColInd_, 
                      B.description_, B.nnz_, B.csrVal_, B.csrRowPtr_, B.csrColInd_,
                      beta,
                      C.description_, C.nnz_, C.csrVal_, C.csrRowPtr_, C.csrColInd_,
                      out.description_, out.csrVal_, out.csrRowPtr_, out.csrColInd_,
                      gemm2Info_, buffer_);
    out.checkError_("hipsparseDcsrgemm2() in CusparseManager::gemm2()");
    return out;
}



CudaMatrix CusparseManager::gemm(const CudaMatrix& lhs, const CudaMatrix& rhs)
{
// Create an empty matrix. Need to set rows, cols, nnz, and allocate arrays!
    CudaMatrix out;
    // Legal matrix sizes depend on whether the matrices are transposed or not!
    int innerSize = out.confirmMultSize(lhs, rhs);

    // Addition in two steps
    //    1) Find nonzero pattern of output
    //    2) Multiply matrices.

    // 1) Find nonzero pattern of output
    // Allocate rowPtr:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrRowPtr_, (out.rows_+1)*sizeof(int));
    out.checkError_("hipMalloc(out.csrRowPtr_) in CusparseManager::gemm()");

    // The following code for finding number of non-zeros is
    // taken from the Nvidia cusparse documentation, section 9.2
    // Only additions are the error checking.
    int *nnzTotalDevHostPtr = &out.nnz_;
    out.sparseStatus_ = hipsparseXcsrgemmNnz( cusparseHandle_, 
                         lhs.operation_, rhs.operation_,
                         out.rows_, out.cols_, innerSize,
                         lhs.description_, lhs.nnz_,
                         lhs.csrRowPtr_, lhs.csrColInd_,
                         rhs.description_, rhs.nnz_,
                         rhs.csrRowPtr_, rhs.csrColInd_,
                         out.description_,
                         out.csrRowPtr_, nnzTotalDevHostPtr);
    out.checkError_("hipsparseXcsrgemmNnz() in CusparseManager::gemm()");
    if ( nnzTotalDevHostPtr != NULL ) {
        out.nnz_ = *nnzTotalDevHostPtr;
    } else {
        int baseC;
        out.cudaStatus_ = hipMemcpy(&out.nnz_, out.csrRowPtr_ + out.rows_,
                         sizeof(int), hipMemcpyDeviceToHost);
        out.checkError_("hipMemcpy(out.csrRowPtr_ + out.rows_) in CusparseManager::gemm()");
        out.cudaStatus_ = hipMemcpy(&baseC, out.csrRowPtr_, sizeof(int),
                         hipMemcpyDeviceToHost);
        out.checkError_("hipMemcpy(baseC) in CusparseManager::gemm()");
        out.nnz_ -= baseC;
    }

     // Allocate the other two arrays:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrVal_, out.nnz_*sizeof(double));
    out.checkError_("hipMalloc(out.csrVal_) in CusparseManager::gemm()");
    out.cudaStatus_ = hipMalloc( (void**)&out.csrColInd_, out.nnz_*sizeof(int));
    out.checkError_("hipMalloc(out.csrColInd_) in CusparseManager::gemm()");
    
    // 2) Multiply the matrices:
    out.sparseStatus_ = hipsparseDcsrgemm(cusparseHandle_,
                     lhs.operation_, rhs.operation_,
                     out.rows_, out.cols_, innerSize,
                     lhs.description_, lhs.nnz_,
                     lhs.csrVal_, lhs.csrRowPtr_, lhs.csrColInd_,
                     rhs.description_, rhs.nnz_,
                     rhs.csrVal_, rhs.csrRowPtr_, rhs.csrColInd_,
                     out.description_,
                     out.csrVal_, out.csrRowPtr_, out.csrColInd_);
    out.checkError_("hipsparseDcsrgemm() in CusparseManager::gemm()");
    
    return out;
}


CudaMatrix CusparseManager::geam(const CudaMatrix& lhs, const CudaMatrix& rhs, const double* alpha, const double* beta)
{
    // Create an empty matrix. Need to set rows, cols, nnz, and allocate arrays!
    CudaMatrix out;
    out.rows_ = lhs.rows_;
    out.cols_ = lhs.cols_;

    // Addition in two steps
    //    1) Find nonzero pattern of output
    //    2) Add matrices.

    // 1) Find nonzero pattern:
    // Allocate rowPtr:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrRowPtr_, (out.rows_+1)*sizeof(int));
    out.checkError_("hipMalloc(out.csrRowPtr_) in CusparseManager::geam()");

    int *nnzTotalDevHostPtr = &out.nnz_;
    out.sparseStatus_ = hipsparseXcsrgeamNnz( cusparseHandle_, out.rows_, out.cols_,
                         lhs.description_, lhs.nnz_,
                         lhs.csrRowPtr_, lhs.csrColInd_,
                         rhs.description_, rhs.nnz_,
                         rhs.csrRowPtr_, rhs.csrColInd_,
                         out.description_, out.csrRowPtr_,
                         nnzTotalDevHostPtr);
    out.checkError_("hipsparseXcsrgeamNnz() in CusparseManager::geam()");
    if ( nnzTotalDevHostPtr != NULL) {
        out.nnz_ = *nnzTotalDevHostPtr;
    } else {
        out.cudaStatus_ = hipMemcpy( &out.nnz_, out.csrRowPtr_ + out.rows_,
                                      sizeof(int), hipMemcpyDeviceToHost);
        out.checkError_("hipMemcpy(out.csrRowPtr_ + rows_) in CusparseManager::geam()");
        int baseC;
        out.cudaStatus_ = hipMemcpy( &baseC, out.csrRowPtr_, sizeof(int),
                                      hipMemcpyDeviceToHost);
        out.checkError_("hipMemcpy(&baseC) in CusparseManager::geam()");
        out.nnz_ -= baseC;
    }

    // Allocate the other two arrays:
    out.cudaStatus_ = hipMalloc( (void**)&out.csrVal_, out.nnz_*sizeof(double));
    out.checkError_("hipMalloc(out.csrVal_) in CusparseManager::geam()");
    out.cudaStatus_ = hipMalloc( (void**)&out.csrColInd_, out.nnz_*sizeof(int));
    out.checkError_("hipMalloc(out.csrColInd_) in CusparseManager::geam()");
    
    // 2) Add matrices
    // Need to create alpha and beta:
    out.sparseStatus_ = hipsparseDcsrgeam(cusparseHandle_, out.rows_, out.cols_,
                     alpha,
                     lhs.description_, lhs.nnz_,
                     lhs.csrVal_, lhs.csrRowPtr_, lhs.csrColInd_,
                     beta,
                     rhs.description_, rhs.nnz_,
                     rhs.csrVal_, rhs.csrRowPtr_, rhs.csrColInd_,
                     out.description_,
                     out.csrVal_, out.csrRowPtr_, out.csrColInd_);
    out.checkError_("hipsparseDcsrgeam() in CusparseManager::geam()");

    return out;
}

CudaMatrix CusparseManager::precond_ilu(const CudaMatrix& A)
{
    CudaMatrix out = A;
    cusparseSolveAnalysisInfo_t analysisInfo;
    cusparseCreateSolveAnalysisInfo(&analysisInfo);
    out.sparseStatus_ = cusparseDcsrsv_analysis(cusparseHandle_, 
                        out.operation_,
                        out.rows_, 
                        out.nnz_, 
                        out.description_,
                        out.csrVal_, 
                        out.csrRowPtr_,
                        out.csrColInd_, 
                        analysisInfo);
    out.checkError_("cusparseDcsrsv_analysis() in CusparseManager::precond_ilu()");
    hipDeviceSynchronize();
    out.sparseStatus_ = cusparseDcsrilu0(cusparseHandle_,
                 out.operation_, 
                 out.rows_, 
                 out.description_, 
                 out.csrVal_,
                 out.csrRowPtr_, 
                 out.csrColInd_,  
                 analysisInfo);
    hipDeviceSynchronize();
    out.checkError_("cusparseDcsrilu0() in CusparseManager::precond_ilu()");
    cusparseDestroySolveAnalysisInfo(analysisInfo);
    return out;
}

CudaArray CusparseManager::biCGStab_ILU_public(const CudaMatrix& A, const int maxit, const CudaArray& x, const double tol)
{
    return instance().biCGStab_ILU(A,maxit,x,tol);
}


CudaArray CusparseManager::biCGStab_ILU(const CudaMatrix& A, const int maxit, const CudaArray& x_in, const double tol)
{
    std::cout << 1 << std::endl;
    CudaMatrix m = A;
    CudaArray x_out = x_in;

    cusparseSolveAnalysisInfo_t analysisInfo_u;
    cusparseSolveAnalysisInfo_t analysisInfo_l;
    cusparseCreateSolveAnalysisInfo(&analysisInfo_u);
    cusparseCreateSolveAnalysisInfo(&analysisInfo_l);
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    std::cout << 2 << std::endl;
    double rho, rhop, beta, alpha, negalpha, omega, negomega, temp, temp2;
    double nrmr, nrmr0;
    rho = 0.0;
    double zero = 0.0;
    double one  = 1.0;
    double mone = -1.0;
    int i = 0;
    int j = 0;
    int n = A.rows_;
    int nnz = A.nnz_;
    std::cout << 3 << std::endl;
    double* r = 0;
    double* t = 0;
    double* s = 0;
    double* rw = 0;
    double* p = 0;
    double* x = x_out.data();
    double* f = 0;
    double* pw = 0;
    double* v = 0;

    std::cout << 4 << std::endl;
    hipMalloc(&r, n*sizeof(double));
    hipMalloc(&t, n*sizeof(double));
    hipMalloc(&s, n*sizeof(double));
    hipMalloc(&rw, n*sizeof(double));
    hipMalloc(&p, n*sizeof(double));
    hipMalloc(&f, n*sizeof(double));
    hipMalloc(&v, n*sizeof(double));
    hipMalloc(&pw, n*sizeof(double));


    thrust::fill(thrust::device, r, r+n, 0.0);
    thrust::fill(thrust::device, t, t+n, 0.0);
    thrust::fill(thrust::device, s, s+n, 0.0);
    thrust::fill(thrust::device, rw, rw+n, 0.0);
    thrust::fill(thrust::device, p, p+n, 0.0);
    thrust::fill(thrust::device, f, f+n, 0.0);
    thrust::fill(thrust::device, v, v+n, 0.0);
    thrust::fill(thrust::device, pw, pw+n, 0.0);
    hipDeviceSynchronize();

    std::cout << 7 << std::endl;
    hipsparseSetMatFillMode(m.description_,HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(m.description_,HIPSPARSE_DIAG_TYPE_UNIT);
    cusparseDcsrsv_analysis(cusparseHandle_,HIPSPARSE_OPERATION_NON_TRANSPOSE,m.rows_,nnz,m.description_,m.csrVal_,m.csrRowPtr_,m.csrColInd_,analysisInfo_l);
    hipDeviceSynchronize();

    std::cout << 8 << std::endl;
    hipsparseSetMatFillMode(m.description_,HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(m.description_,HIPSPARSE_DIAG_TYPE_NON_UNIT);
    cusparseDcsrsv_analysis(cusparseHandle_,HIPSPARSE_OPERATION_NON_TRANSPOSE,m.rows_,nnz,m.description_,m.csrVal_,m.csrRowPtr_,m.csrColInd_,analysisInfo_u);
    hipDeviceSynchronize();
    std::cout << 9 << std::endl;
    m.sparseStatus_ = cusparseDcsrilu0(cusparseHandle_,
                 m.operation_, 
                 m.rows_, 
                 m.description_, 
                 m.csrVal_,
                 m.csrRowPtr_, 
                 m.csrColInd_,  
                 analysisInfo_l);
    hipDeviceSynchronize();
    std::cout << A << std::endl;
    std::cout << m << std::endl;
    m.checkError_("cusparseDcsrilu0() in CusparseManager::precond_ilu()");
    std::cout << 10 << std::endl;
    //compute initial residual r0=b-Ax0 (using initial guess in x)

    // Residual r er output. Linjene under er -Ax0
    hipsparseDcsrmv(cusparseHandle_, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, A.description_, A.csrVal_, A.csrRowPtr_, A.csrColInd_, x, &zero, r);
    hipblasDscal(cublasHandle, n, &mone, r, 1);
    hipblasDaxpy(cublasHandle, n, &one, f, 1, r, 1);
    std::cout << 11 << std::endl;
    //copy residual r into r^{\hat} and p
    hipblasDcopy(cublasHandle, n, r, 1, rw, 1);
    hipblasDcopy(cublasHandle, n, r, 1, p, 1); 
    hipblasDnrm2(cublasHandle, n, r, 1, &nrmr0);
    std::cout << 12 << std::endl;
    for (i=0; i<maxit; ){
        rhop = rho;
        hipblasDdot(cublasHandle, n, rw, 1, r, 1, &rho);

        if (i > 0){
            beta= (rho/rhop) * (alpha/omega);
            negomega = -omega;
            hipblasDaxpy(cublasHandle,n, &negomega, v, 1, p, 1);
            hipblasDscal(cublasHandle,n, &beta, p, 1);
            hipblasDaxpy(cublasHandle,n, &one, r, 1, p, 1);
        }
        //preconditioning step (lower and upper triangular solve)

        hipsparseSetMatFillMode(m.description_,HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(m.description_,HIPSPARSE_DIAG_TYPE_UNIT);
        cusparseDcsrsv_solve(cusparseHandle_,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,&one,m.description_,m.csrVal_,m.csrRowPtr_,m.csrColInd_,analysisInfo_l,p,t);

        hipsparseSetMatFillMode(m.description_,HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(m.description_,HIPSPARSE_DIAG_TYPE_NON_UNIT);
        cusparseDcsrsv_solve(cusparseHandle_,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,&one,m.description_,m.csrVal_,m.csrRowPtr_,m.csrColInd_,analysisInfo_u,t,pw);


        //matrix-vector multiplication

        hipsparseDcsrmv(cusparseHandle_, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, A.description_, A.csrVal_, A.csrRowPtr_, A.csrColInd_, pw, &zero, v);

        hipblasDdot(cublasHandle,n, rw, 1, v, 1,&temp);
        alpha= rho / temp;
        negalpha = -(alpha);
        hipblasDaxpy(cublasHandle,n, &negalpha, v, 1, r, 1);
        hipblasDaxpy(cublasHandle,n, &alpha,    pw, 1, x, 1);
        hipblasDnrm2(cublasHandle, n, r, 1, &nrmr);

        if (nrmr < tol*nrmr0){
            j=5;
            break;
        }

        //preconditioning step (lower and upper triangular solve)
        hipsparseSetMatFillMode(m.description_,HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(m.description_,HIPSPARSE_DIAG_TYPE_UNIT);
        cusparseDcsrsv_solve(cusparseHandle_,HIPSPARSE_OPERATION_NON_TRANSPOSE,n, &one,m.description_,m.csrVal_,m.csrRowPtr_,m.csrColInd_,analysisInfo_l,r,t);

        hipsparseSetMatFillMode(m.description_,HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(m.description_,HIPSPARSE_DIAG_TYPE_NON_UNIT);
        cusparseDcsrsv_solve(cusparseHandle_,HIPSPARSE_OPERATION_NON_TRANSPOSE,n, &one,m.description_,m.csrVal_,m.csrRowPtr_,m.csrColInd_,analysisInfo_u,t,s);

        //matrix-vector multiplication

        hipsparseDcsrmv(cusparseHandle_, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, A.description_, A.csrVal_, A.csrRowPtr_, A.csrColInd_, s, &zero, t);

        hipblasDdot(cublasHandle,n, t, 1, r, 1,&temp);
        hipblasDdot(cublasHandle,n, t, 1, t, 1,&temp2);
        omega = temp / temp2;
        negomega = -(omega);
        hipblasDaxpy(cublasHandle,n, &omega, s, 1, x, 1);
        hipblasDaxpy(cublasHandle,n, &negomega, t, 1, r, 1);

        hipblasDnrm2(cublasHandle,n, r, 1,&nrmr);

        if (nrmr < tol*nrmr0){
            i++;
            j=0;
            break;
        }
        i++;
    }  
    cusparseDestroySolveAnalysisInfo(analysisInfo_u);
    cusparseDestroySolveAnalysisInfo(analysisInfo_l);
    hipFree(r);
    hipFree(t);
    hipFree(s);
    hipFree(rw);
    hipFree(p);
    hipFree(f);
    hipFree(v);
    hipFree(pw);
    hipblasDestroy(cublasHandle);

    return x_out;
    /*CudaMatrix out = instance.precondILU(A);

    for (i=0; i<maxit; ){
        rhop = rho;
        checkCudaErrors(hipblasDdot(cublasHandle, n, rw, 1, r, 1, &rho));

        if (i > 0){
            beta= (rho/rhop) * (alpha/omega);
            negomega = -omega;
            checkCudaErrors(hipblasDaxpy(cublasHandle,n, &negomega, v, 1, p, 1));
            checkCudaErrors(hipblasDscal(cublasHandle,n, &beta, p, 1));
            checkCudaErrors(hipblasDaxpy(cublasHandle,n, &one, r, 1, p, 1));
        }

        checkCudaErrors(hipblasDdot(cublasHandle,n, rw, 1, v, 1,&temp));
        alpha= rho / temp;
        negalpha = -(alpha);
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &negalpha, v, 1, r, 1));
        checkCudaErrors(hipblasDaxpy(cublasHandle,n, &alpha,        pw, 1, x, 1));
        checkCudaErrors(hipblasDnrm2(cublasHandle, n, r, 1, &nrmr));

        if (nrmr < tol*nrmr0){
            j=5;
            break;
        }

        hipblasDdot(cublasHandle,n, t, 1, r, 1,&temp);
        hipblasDdot(cublasHandle,n, t, 1, t, 1,&temp2);
        omega= temp / temp2;
        negomega = -(omega);
        hipblasDaxpy(cublasHandle,n, &omega, s, 1, x, 1);
        hipblasDaxpy(cublasHandle,n, &negomega, t, 1, r, 1);

        hipblasDnrm2(cublasHandle,n, r, 1,&nrmr);

        if (nrmr < tol*nrmr0){
            i++;
            j=0;
            break;
        }
        i++;
    }*/
}

CudaMatrix CusparseManager::precondILU(const CudaMatrix& A)
{
    return instance().precond_ilu(A);
}